#include "hip/hip_runtime.h"
﻿/*
 *   Copyright 2022 The Regents of the University of California, Davis
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 */

#include <hip/hip_runtime_api.h>
#include <gpu_btree.h>
#include <stdlib.h>
#include <thrust/sequence.h>
#include <algorithm>
#include <cmd.hpp>
#include <cstdint>
#include <gpu_timer.hpp>
#include <numeric>
#include <random>
#include <rkg.hpp>
#include <string>
#include <unordered_set>
#include <validation.hpp>
#include <vector>

#include <device_bump_allocator.hpp>
#include <slab_alloc.hpp>

template <typename BTree,
          bool supportsVersioning,
          typename KeyT,
          typename ValueT,
          typename PairT,
          typename SetT,
          typename SizeT,
          typename Function0,
          typename Function1>
std::vector<std::pair<uint32_t, uint32_t>> bench_versioned_insert_range(
    thrust::device_vector<KeyT> &d_keys,
    thrust::device_vector<ValueT> &d_values,
    SizeT initial_tree_size,
    thrust::device_vector<KeyT> &d_lower_bound,
    thrust::device_vector<KeyT> &d_upper_bound,
    thrust::device_vector<PairT> &d_results,
    SizeT average_range_length,
    std::vector<KeyT> &h_keys,
    std::vector<KeyT> &h_lower_bound,
    bool in_place,
    bool validate_result,
    bool validate_tree_structure,
    const SetT &ref_set_v0,
    const SetT &ref_set_v1,
    std::size_t num_experiments,
    Function0 &to_value,
    Function1 &to_upper_bound) {
  (void)in_place;
  (void)ref_set_v0;
  (void)ref_set_v1;
  hipStream_t insertion_stream{0};
  hipStream_t concurrent_ops_stream{0};
  float average_insertion_seconds(0.0f);
  float average_ops_seconds(0.0f);

  const KeyT invalid_key     = std::numeric_limits<KeyT>::max();
  const ValueT invalid_value = std::numeric_limits<ValueT>::max();
  const PairT invalid_pair(invalid_key, invalid_value);

  auto second_insertion_batch_size = d_keys.size() - initial_tree_size;

  std::vector<std::pair<uint32_t, uint32_t>> epochs_usage;

  for (std::size_t exp = 0; exp < num_experiments; exp++) {
    std::cout << "Experiment " << exp << "/" << num_experiments << "...";
    std::cout << std::endl;

    BTree tree;
    gpu_timer insert_timer(insertion_stream);
    insert_timer.start_timer();
    if constexpr (supportsVersioning) {
      tree.insert(d_keys.data().get(),
                  d_values.data().get(),
                  initial_tree_size,
                  insertion_stream,
                  in_place);
    } else {
      tree.insert(d_keys.data().get(), d_values.data().get(), initial_tree_size, insertion_stream);
    }
    insert_timer.stop_timer();
    cuda_try(hipDeviceSynchronize());
    average_insertion_seconds += insert_timer.get_elapsed_s();

    if (validate_tree_structure) {
      std::vector<KeyT> h_keys_v0(h_keys.begin(), h_keys.begin() + initial_tree_size);
      tree.validate_tree_structure(h_keys_v0, to_value);
      std::cout << "Validation success @v0" << std::endl;
    }

    cuda_try(hipProfilerStart());

    gpu_timer ops_timer(concurrent_ops_stream);
    ops_timer.start_timer();
    tree.concurrent_insert_range(d_keys.data().get() + initial_tree_size,
                                 d_values.data().get() + initial_tree_size,
                                 second_insertion_batch_size,
                                 d_lower_bound.data().get(),
                                 d_upper_bound.data().get(),
                                 d_lower_bound.size(),
                                 d_results.data().get(),
                                 average_range_length,
                                 concurrent_ops_stream);
    ops_timer.stop_timer();
    cuda_try(hipDeviceSynchronize());
    average_ops_seconds += ops_timer.get_elapsed_s();
    cuda_try(hipProfilerStop());

    epochs_usage = tree.compute_reclaimer_stats();

    if (validate_result) {
      std::cout << "Validating... ";
      utils::validate_concurrent_ops(h_lower_bound,
                                     d_results,
                                     average_range_length,
                                     ref_set_v0,
                                     ref_set_v1,
                                     to_value,
                                     to_upper_bound);
      thrust::fill(thrust::device, d_results.begin(), d_results.end(), invalid_pair);
    }
    if (validate_tree_structure) {
      std::vector<KeyT> h_keys_v1(h_keys.begin(), h_keys.end());
      tree.validate_tree_structure(h_keys_v1, to_value);
      std::cout << "Validation success @v1" << std::endl;
    }
  }

  average_insertion_seconds /= float(num_experiments);
  average_ops_seconds /= float(num_experiments);

  float insertion_rate = float(initial_tree_size) / 1e6 / average_insertion_seconds;
  float ops_rate =
      float(second_insertion_batch_size + d_lower_bound.size()) / 1e6 / average_ops_seconds;

  std::cout << "build_rate: " << insertion_rate << std::endl;
  std::cout << "concurrent_ops_rate: " << ops_rate << std::endl;

  return epochs_usage;
}

int main(int argc, char **argv) {
  auto arguments = std::vector<std::string>(argv, argv + argc);

  uint32_t initial_tree_size =
      get_arg_value<uint32_t>(arguments, "initial-size").value_or(1'000'000);
  uint32_t num_operations       = get_arg_value<uint32_t>(arguments, "num-ops").value_or(1'000'000);
  float update_ratio            = get_arg_value<float>(arguments, "update-ratio").value_or(0.5f);
  uint32_t average_range_length = get_arg_value<uint32_t>(arguments, "range-length").value_or(16);

  int device_id = get_arg_value<int>(arguments, "device").value_or(0);
  std::size_t num_experiments =
      get_arg_value<std::size_t>(arguments, "num-experiments").value_or(1llu);
  bool validate        = get_arg_value<bool>(arguments, "validate").value_or(false);
  bool validate_result = get_arg_value<bool>(arguments, "validate-result").value_or(validate);
  bool validate_tree   = get_arg_value<bool>(arguments, "validate-tree").value_or(validate);

  std::string output_dir = get_arg_value<std::string>(arguments, "output-dir").value_or("./");

  int device_count;
  hipGetDeviceCount(&device_count);
  hipDeviceProp_t devProp;
  if (device_id < device_count) {
    hipSetDevice(device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device[" << device_id << "]: " << devProp.name << std::endl;
  } else {
    std::cout << "No capable CUDA device found." << std::endl;
    std::terminate();
  }

  std::string device_name(devProp.name);
  std::replace(device_name.begin(), device_name.end(), ' ', '-');

  uint32_t num_insertions  = num_operations * update_ratio;
  uint32_t num_range_query = num_operations - num_insertions;
  uint32_t num_keys        = num_insertions + initial_tree_size;

  std::cout << "Benchmarking...\n";
  std::cout << std::boolalpha;
  std::cout << "initial_tree_size = " << initial_tree_size << ",\n";
  std::cout << "num_operations = " << num_operations << ", ";
  std::cout << "num_insertions = " << num_insertions << ", ";
  std::cout << "num_range_query = " << num_range_query << ", ";
  std::cout << "average_range_length = " << average_range_length << "\n";
  std::cout << "update_ratio = " << update_ratio << ", ";
  std::cout << "num_experiments = " << num_experiments << ", \n";
  std::cout << "validate_tree = " << validate_tree << ", ";
  std::cout << "validate_result = " << validate_result << "\n";

  std::cout << "------------------------\n";
  std::cout << "Generating input...\n";

  using key_type                 = uint32_t;
  using value_type               = uint32_t;
  using pair                     = pair<key_type, value_type>;
  const key_type invalid_key     = std::numeric_limits<key_type>::max();
  const value_type invalid_value = std::numeric_limits<value_type>::max();
  const pair invalid_pair(invalid_key, invalid_value);
  auto to_value       = [] __host__ __device__(key_type x) { return x % 10; };
  auto to_upper_bound = [average_range_length] __host__ __device__(key_type x) {
    return x + average_range_length;
  };

  unsigned seed = 0;
  std::random_device rd;
  std::mt19937_64 rng(seed);
  auto d_keys   = thrust::device_vector<key_type>(num_keys, invalid_key);
  auto d_values = thrust::device_vector<value_type>(num_keys, invalid_value);

  auto d_range_lower = thrust::device_vector<key_type>(num_range_query, invalid_key);
  auto d_range_upper = thrust::device_vector<key_type>(num_range_query, invalid_key);
  auto d_range_results =
      thrust::device_vector<pair>(num_range_query * average_range_length, invalid_pair);

  // host vectors
  auto h_keys = rkg::generate_keys<key_type>(num_keys, rng, rkg::distribution_type::unique_random);
  auto h_range_lower = std::vector<key_type>(num_range_query, invalid_key);

  rkg::prep_experiment_range_query(h_keys, initial_tree_size, h_range_lower, num_range_query, rng);

  // copy to device
  d_keys        = h_keys;
  d_range_lower = h_range_lower;

  // assign values and upper bound
  thrust::transform(thrust::device, d_keys.begin(), d_keys.end(), d_values.begin(), to_value);
  thrust::transform(thrust::device,
                    d_range_lower.begin(),
                    d_range_lower.end(),
                    d_range_upper.begin(),
                    to_upper_bound);

  std::set<key_type> cpu_ref_set_v0;  // contains initial tree
  std::set<key_type> cpu_ref_set_v1;  // contains inserted keys
  if (validate_result) {
    std::cout << "Building CPU reference sets...\n";
    cpu_ref_set_v0.insert(h_keys.begin(), h_keys.begin() + initial_tree_size);
    cpu_ref_set_v1.insert(h_keys.begin() + initial_tree_size, h_keys.end());
  }

  static constexpr int branching_factor = 16;
  using node_type           = GpuBTree::node_type<key_type, value_type, branching_factor>;
  using slab_allocator_type = device_allocator::SlabAllocLight<node_type, 8, 128 * 64, 16, 128>;
  using bump_allocator_type = device_bump_allocator<node_type>;

  using blink_tree_slab_type =
      GpuBTree::gpu_blink_tree<key_type, value_type, branching_factor, slab_allocator_type>;
  using blink_tree_bump_type =
      GpuBTree::gpu_blink_tree<key_type, value_type, branching_factor, bump_allocator_type>;

  using vblink_tree_slab_type =
      GpuBTree::gpu_versioned_btree<key_type, value_type, branching_factor, slab_allocator_type>;
  using vblink_tree_bump_type =
      GpuBTree::gpu_versioned_btree<key_type, value_type, branching_factor, bump_allocator_type>;

  std::string report_dir = output_dir + '/' + device_name + "/versioned_insert_range/";
  std::filesystem::create_directories(report_dir);

  std::string filename = "memory_" +
                         std::to_string(static_cast<unsigned>(initial_tree_size / 1e6)) + "m_" +
                         std::to_string(static_cast<unsigned>(num_operations / 1e6)) + "m_" +
                         std::to_string(static_cast<unsigned>(update_ratio * 100.0f)) + "_" +
                         std::to_string(average_range_length) + ".csv";

  std::fstream result_output(report_dir + filename, std::ios::out);
  result_output << "initial_tree_size" << ',';
  result_output << "num_insertions" << ',';
  result_output << "num_range_query" << ',';
  result_output << "num_experiments" << ',';
  result_output << "epoch_number" << ',';
  result_output << "pointers_count" << ',';
  result_output << "pointers_allocated_total" << ',';
  result_output << "ratio" << ',';
  result_output << "tree_gibs" << ',';

  result_output << '\n';

  std::cout << "Running experiment...\n";
  {
    auto epoch_usage =
        bench_versioned_insert_range<vblink_tree_slab_type, true>(d_keys,
                                                                  d_values,
                                                                  initial_tree_size,
                                                                  d_range_lower,
                                                                  d_range_upper,
                                                                  d_range_results,
                                                                  average_range_length,
                                                                  h_keys,
                                                                  h_range_lower,
                                                                  true,
                                                                  validate_result,
                                                                  validate_tree,
                                                                  cpu_ref_set_v0,
                                                                  cpu_ref_set_v1,
                                                                  num_experiments,
                                                                  to_value,
                                                                  to_upper_bound);

    auto to_gibs = (1ull << 30);
    for (std::size_t i = 0; i < epoch_usage.size(); i++) {
      result_output << initial_tree_size << ',';
      result_output << num_insertions << ',';
      result_output << num_range_query << ',';
      result_output << num_experiments << ',';
      result_output << i << ',';
      result_output << epoch_usage[i].first << ',';
      result_output << epoch_usage[i].second << ',';
      result_output << double(epoch_usage[i].first) / epoch_usage[i].second << ',';
      result_output << double(epoch_usage[i].first * 128.0) / double(to_gibs) << ",\n";
    }
  }
}
